#include "hip/hip_runtime.h"
/*
 * LSTMNetwork.cpp
 *
 *  Created on: Jul 27, 2016
 *      Author: trabucco
 */

#include "LSTMNetwork.cuh"

__global__ void forwardPass(Neuron **neurons, double *connections, double *activations, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			activations[idx] = neurons[idx]->forward(connections);
			//printf("F Neuron %d : %f\n", size, activations[idx]);
		}
	}
}

__global__ void backwardPass(Neuron **neurons, double *weightedError, double *errorSum, double learningRate, int connections, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			double *contribution = neurons[idx]->backward(weightedError[idx], learningRate);
			//printf("B Neurons %d\n", size);
			for (int j = 0; j < connections; j++) {
				errorSum[j] += contribution[j];
			}
		}
	}
}

__global__ void forwardPassLSTM(MemoryBlock **blocks, double *connections, double *activations, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			double *blockActivation = blocks[idx]->forward(connections);
			//printf("F Cells %d\n", blocks[idx]->nCells);
			for (int j = 0; j < blocks[i]->nCells; j++) activations[idx * blocks[i]->nCells + j] = blockActivation[j];
		}
	}
}

__global__ void backwardPassLSTM(MemoryBlock **blocks, double **weightedError, double *errorSum, double learningRate, int connections, int size, int cycles) {
	int maxId = gridDim.x * blockDim.x;
	for (int i = 0; i < (cycles); i++) {
		int idx = (threadIdx.x + blockIdx.x * blockDim.x) + (maxId * i);
		if (idx < size) {
			if (idx == 0) printf("B Cells %p\n", blocks[idx]);
			double *contribution = blocks[idx]->backward(weightedError[idx], learningRate);
			for (int j = 0; j < connections; j++) {
				errorSum[j] += contribution[j];
			}
		}
	}
}

LSTMNetwork::LSTMNetwork(int is, int b, int c, double l, double d) {
	// TODO Auto-generated constructor stub
	inputSize = is;
	learningRate = l;
	decayRate = d;
	for (int i = 0; i < b; i++) {
		blocks.push_back(MemoryBlock(c, is));
	}
}

LSTMNetwork::~LSTMNetwork() {
	// TODO Auto-generated destructor stub
}

int LSTMNetwork::getPreviousNeurons() {
	return (layers.size() == 0) ? (blocks.size() * blocks[0].nCells) : layers[layers.size() - 1].size();
}

void LSTMNetwork::addLayer(int size) {
	vector<Neuron> buffer;
	for (int i = 0; i < size; i++) {
		buffer.push_back(Neuron(getPreviousNeurons()));
	} layers.push_back(buffer);
}

vector<double> LSTMNetwork::classify(vector<double> input) {
	double *output = (double *)malloc(sizeof(double) * blocks.size() * blocks[0].nCells),
			*connections;
	hipMalloc((void **)&connections, sizeof(double) * input.size());
	hipMemcpy(&connections[0], &input[0], (sizeof(double) * input.size()), hipMemcpyHostToDevice);
	if (input.size() == inputSize) {
		// calculate activations from bottom up
		double *activations;
		hipMalloc((void **)&activations, (sizeof(double) * blocks.size() * blocks[0].nCells));

		MemoryBlock **deviceBlocks, **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
		for (int i = 0; i < blocks.size(); i++) {
			hipMemcpy(&(blocks[i].impulse[0]), &connections[0], (sizeof(double) * blocks[i].nConnections), hipMemcpyDeviceToHost);
		}
		hipMalloc((void **)&deviceBlocks, sizeof(MemoryBlock *) * blocks.size());
		for (int i = 0; i < blocks.size(); i++) {
			MemoryBlock *db = MemoryBlock::copyToGPU(&blocks[i]);
			hipMemcpy(&deviceBlocks[i], &db, sizeof(MemoryBlock *), hipMemcpyHostToDevice);
		} forwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, connections, activations, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
		hipDeviceSynchronize();

		hipMemcpy(&blockBuffer[0], &deviceBlocks[0], (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);
		for (int i = 0; i < blocks.size(); i++) {
			blocks[i] = *MemoryBlock::copyFromGPU(blockBuffer[i]);
		} free(blockBuffer);
		hipFree(deviceBlocks);

		hipFree(connections);
		hipMalloc((void **)&connections, (sizeof(double) * blocks.size() * blocks[0].nCells));
		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * blocks.size() * blocks[0].nCells), hipMemcpyDeviceToDevice);
		hipFree(activations);
		free(output);
		output = (double *)malloc(sizeof(double) * layers[layers.size() - 1].size());

		for (int i = 0; i < layers.size(); i++) {
			hipMalloc((void **)&activations, (sizeof(double) * layers[i].size()));

			Neuron **deviceNeurons, **neuronBuffer = (Neuron **)malloc(sizeof(Neuron *) * layers[i].size());
			for (int j = 0; j < layers[i].size(); j++) {
				hipMemcpy(&(layers[i][j].impulse[0]), &connections[0], (sizeof(double) * layers[i][j].connections), hipMemcpyDeviceToHost);
			}
			hipMalloc((void **)&deviceNeurons, sizeof(Neuron *) * layers[i].size());
			for (int j = 0; j < layers[i].size(); j++) {
				Neuron *dn = Neuron::copyToGPU(&layers[i][j]);
				hipMemcpy(&deviceNeurons[j], &dn, sizeof(Neuron *), hipMemcpyHostToDevice);
			} forwardPass<<<maxBlocks, maxThreads>>>(deviceNeurons, connections, activations, layers[i].size(), ceil((double)layers[i].size() / (double)(maxBlocks * maxThreads)));
			hipDeviceSynchronize();

			hipFree(connections);
			hipMalloc((void **)&connections, (sizeof(double) * layers[i].size()));
			hipMemcpy(&connections[0], &activations[0], (sizeof(double) * layers[i].size()), hipMemcpyDeviceToDevice);
			hipMemcpy(&neuronBuffer[0], &deviceNeurons[0], (sizeof(Neuron *) * layers[i].size()), hipMemcpyDeviceToHost);
			for (int j = 0; j < layers[i].size(); j++) {
				layers[i][j] = *Neuron::copyFromGPU(neuronBuffer[j]);
			} if (i == (layers.size() - 1)) hipMemcpy(&output[0], &activations[0], (sizeof(double) * layers[layers.size() - 1].size()), hipMemcpyDeviceToHost);
			hipFree(activations);
			hipFree(deviceNeurons);
			free(neuronBuffer);
		} vector<double> result(&output[0], &output[layers[layers.size() - 1].size()]);
		free(output);
		hipFree(connections);
		return result;
	} else return vector<double>();
}

vector<double> LSTMNetwork::train(vector<double> input, vector<double> target) {
	Neuron ***deviceNeurons = (Neuron ***)malloc(sizeof(Neuron *) * layers.size());
	double *output = (double *)malloc(blocks.size() * blocks[0].nCells * sizeof(double)),
			*connections;
	hipMalloc((void **)&connections, sizeof(double) * input.size());
	hipMemcpy(&connections[0], &input[0], (sizeof(double) * input.size()), hipMemcpyHostToDevice);
	if (input.size() == inputSize) {
		// start forward pass
		// calculate activations from bottom up
		double *activations;
		hipMalloc((void **)&activations, (sizeof(double) * blocks.size() * blocks[0].nCells));
		MemoryBlock **deviceBlocks;
		for (int i = 0; i < blocks.size(); i++) {
			hipMemcpy(&(blocks[i].impulse[0]), &connections[0], (sizeof(double) * blocks[i].nConnections), hipMemcpyDeviceToHost);
		} hipMalloc((void **)&deviceBlocks, sizeof(MemoryBlock *) * blocks.size());
		for (int i = 0; i < blocks.size(); i++) {
			cout << "Test " << blocks[i].cells[0]->nConnections << endl;
			MemoryBlock *db = MemoryBlock::copyToGPU(&blocks[i]);
			hipMemcpy(&deviceBlocks[i], &db, sizeof(MemoryBlock *), hipMemcpyHostToDevice);
		} forwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, connections, activations, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
		hipDeviceSynchronize();
		hipFree(connections);
		hipMalloc((void **)&connections, (sizeof(double) * blocks.size() * blocks[0].nCells));
		hipMemcpy(&connections[0], &activations[0], (sizeof(double) * blocks.size() * blocks[0].nCells), hipMemcpyDeviceToDevice);
		hipFree(activations);
		free(output);
		output = (double *)malloc(sizeof(double) * layers[layers.size() - 1].size());

		for (int i = 0; i < layers.size(); i++) {
			hipMalloc((void **)&activations, (sizeof(double) * layers[i].size()));

			Neuron **layerNeurons;
			for (int j = 0; j < layers[i].size(); j++) {
				hipMemcpy(&(layers[i][j].impulse[0]), &connections[0], (sizeof(double) * layers[i][j].connections), hipMemcpyDeviceToHost);
			}
			hipMalloc((void **)&layerNeurons, sizeof(Neuron *) * layers[i].size());
			for (int j = 0; j < layers[i].size(); j++) {
				Neuron *dn = Neuron::copyToGPU(&layers[i][j]);
				hipMemcpy(&layerNeurons[j], &dn, sizeof(Neuron *), hipMemcpyHostToDevice);
			} deviceNeurons[i] = layerNeurons;
			forwardPass<<<maxBlocks, maxThreads>>>(layerNeurons, connections, activations, layers[i].size(), ceil((double)layers[i].size() / (double)(maxBlocks * maxThreads)));
			hipDeviceSynchronize();
			hipFree(connections);
			hipMalloc((void **)&connections, (sizeof(double) * layers[i].size()));
			cout << "copy activations " << hipMemcpy(&connections[0], &activations[0], (sizeof(double) * layers[i].size()), hipMemcpyDeviceToDevice);
			hipFree(activations);
		} hipFree(connections);

		// start backward pass
		double *weightedError;
		hipMalloc((void **)&weightedError, (sizeof(double) * layers[layers.size() - 1].size()));
		for (int i = 0; i < layers[layers.size() - 1].size(); i++) {
			double error = (output[i] - target[i]);
			output[i] = error;
			hipMemcpy(&weightedError[i], &error, sizeof(double), hipMemcpyHostToDevice);
		} for (int i = (layers.size() - 1); i >= 0; i--) {
			double *errorSum;
			hipMalloc((void **)&errorSum, (sizeof(double) * layers[i][0].connections));
			hipMemset(&errorSum[0], 0, (sizeof(double) * layers[i][0].connections));

			// compute the gradient
			backwardPass<<<maxBlocks, maxThreads>>>(deviceNeurons[i], weightedError, errorSum, learningRate, layers[i][0].connections, layers[i].size(), ceil((double)layers[i].size() / (double)(maxBlocks * maxThreads)));
			hipDeviceSynchronize();
			hipFree(weightedError);
			hipMalloc((void **)&weightedError, (sizeof(double) * layers[i][0].connections));
			cout << "copy sum " << hipMemcpy(&weightedError[0], &errorSum[0], (sizeof(double) * layers[i][0].connections), hipMemcpyDeviceToDevice);

			Neuron **neuronBuffer = (Neuron **)malloc(sizeof(Neuron) * layers[i].size());
			cout << "copy neurons " << hipMemcpy(&neuronBuffer[0], &deviceNeurons[i][0], (sizeof(Neuron *) * layers[i].size()), hipMemcpyDeviceToHost);
			for (int j = 0; j < layers[i].size(); j++) {
				layers[i][j] = *Neuron::copyFromGPU(neuronBuffer[j]);
			} free(neuronBuffer);
			hipFree(deviceNeurons[i]);
		}
		double **errorChunks, *errorSum;
		hipMalloc((void **)&errorChunks, (sizeof(double *) * blocks.size()));
		hipMalloc((void **)&errorSum, (sizeof(double) * blocks[0].nConnections));
		hipMemset(&errorSum[0], 0.0, (sizeof(double) * blocks[0].nConnections));
		for (int i = 0; i < (blocks.size()); i++) {
			double *chunk;
			hipMalloc((void **)&chunk, (sizeof(double) * blocks[i].nCells));
			hipMemcpy(&chunk[0], &weightedError[(i * blocks[i].nCells)], (sizeof(double) * blocks[i].nCells), hipMemcpyDeviceToDevice);
			hipMemcpy(&errorChunks[i], &chunk, (sizeof(double *)), hipMemcpyHostToDevice);
		} backwardPassLSTM<<<maxBlocks, maxThreads>>>(deviceBlocks, errorChunks, errorSum, learningRate, blocks[0].nConnections, blocks.size(), ceil((double)blocks.size() / (double)(maxBlocks * maxThreads)));
		hipDeviceSynchronize();

		MemoryBlock **blockBuffer = (MemoryBlock **)malloc(sizeof(MemoryBlock *) * blocks.size());
		cout << blocks.size() << " copy blocks " << hipMemcpy(blockBuffer, deviceBlocks, (sizeof(MemoryBlock *) * blocks.size()), hipMemcpyDeviceToHost);

		cout << "CB  " << blockBuffer[0] << endl;

		for (int i = 0; i < blocks.size(); i++) {
			MemoryBlock temp = *MemoryBlock::copyFromGPU(blockBuffer[i]);
			blocks[i] = temp;
			cout << "Test copy " << blocks[i].cells[0]->nConnections << endl;
		}

		hipFree(deviceBlocks);
		free(deviceNeurons);
		hipFree(weightedError);
		hipFree(errorChunks);
		hipFree(errorSum);

		learningRate *= decayRate;
		vector<double> result(&output[0], &output[layers[layers.size() - 1].size()]);
		free(output);
		return result;
	} else {
		cout << "Target size mismatch" << endl;
		return vector<double>();
	}
}
