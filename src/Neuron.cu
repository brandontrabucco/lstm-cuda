#include "hip/hip_runtime.h"
/*
 * Neuron.cpp
 *
 *  Created on: Jun 22, 2016
 *      Author: trabucco
 */

#include "Neuron.cuh"

long long Neuron::n = 0;

Neuron::Neuron(int nConnections) {
	// TODO Auto-generated constructor stub
	activation = 0; activationPrime = 0;
	connections = nConnections;
	default_random_engine g(time(0) + (n++));
	normal_distribution<double> d(0, 1);
	weightedError = (double *)malloc(sizeof(double) * nConnections);
	weight = (double *)malloc(sizeof(double) * nConnections);
	impulse = (double *)calloc(nConnections, sizeof(double));
	for (int i = 0; i < connections; i++) {
		weight[i] = (d(g));
	}
}

Neuron::~Neuron() {
	// TODO Auto-generated destructor stub
}

__device__ double Neuron::sigmoid(double input) {
	return 1 / (1 + exp(-input));
}

__device__ double Neuron::sigmoidPrime(double input) {
	return sigmoid(input) * (1 - sigmoid(input));
}

__device__ double Neuron::activate(double input) {
	return tanh(input);
}

__device__ double Neuron::activatePrime(double input) {
	return (1 - (tanh(input) * tanh(input)));
}

__device__ double Neuron::forward(double *input) {
	double sum = 0;
	// find the weighted sum of all input
	for (int i = 0; i < connections; i++) {
		//cout << weight[i] << " ";
		//sum += input[i] * weight[i];
		sum += input[i];
		//sum += weight[i];
	}// cout << " sum : " << sum << " weights : " << weight.size() << endl;
	activation = activate(sum);
	activationPrime = activatePrime(sum);
	return activation;
}

__device__ double *Neuron::backward(double errorPrime, double learningRate) {
	// update all weights
	for (int i = 0; i < connections; i++) {
		weightedError[i] = (errorPrime * weight[i] * activationPrime);
		weight[i] -= learningRate * errorPrime * impulse[i];
	}
	return weightedError;
}

Neuron *Neuron::copyToGPU(Neuron *data) {
	Neuron *neuron;
	hipMalloc((void **)&neuron, (sizeof(Neuron)));
	hipDeviceSynchronize();
	hipMemcpy(neuron, data, sizeof(Neuron), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	double *e;
	double *w;
	double *i;
	hipMalloc((void **)&e, (sizeof(double) * data->connections));
	hipMalloc((void **)&w, (sizeof(double) * data->connections));
	hipMalloc((void **)&i, (sizeof(double) * data->connections));
	hipDeviceSynchronize();

	hipMemcpy(e, data->weightedError, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipMemcpy(w, data->weight, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipMemcpy(i, data->impulse, (sizeof(double) * data->connections), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipMemcpy(&(neuron->weightedError), &e, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(neuron->weight), &w, sizeof(double *), hipMemcpyHostToDevice);
	hipMemcpy(&(neuron->impulse), &i, sizeof(double *), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	return neuron;
}

Neuron *Neuron::copyFromGPU(Neuron *data) {
	Neuron *neuron;
	neuron = (Neuron *)malloc((sizeof(Neuron)));
	hipDeviceSynchronize();
	hipMemcpy(neuron, data, sizeof(Neuron), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	double *e;
	double *w;
	double *i;
	e = (double *)malloc(sizeof(double) * neuron->connections);
	w = (double *)malloc(sizeof(double) * neuron->connections);
	i = (double *)malloc(sizeof(double) * neuron->connections);
	hipDeviceSynchronize();

	hipMemcpy(e, neuron->weightedError, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipMemcpy(w, neuron->weight, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipMemcpy(i, neuron->impulse, (sizeof(double) * neuron->connections), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();

	memcpy(&(neuron->weightedError), &e, sizeof(double *));
	memcpy(&(neuron->weight), &w, sizeof(double *));
	memcpy(&(neuron->impulse), &i, sizeof(double *));
	hipDeviceSynchronize();

	return neuron;
}

